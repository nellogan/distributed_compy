#include "hip/hip_runtime.h"
#include "GPUReductionSum.h"

template <class T>
struct SharedMemory {
  __device__ inline operator T *() {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }

  __device__ inline operator const T *() const {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }
};

template <typename T, unsigned int block_size, bool n_is_pow2>
__global__ void SumKernel(T *g_idata, T *g_odata, unsigned int n) {
    T *sdata = SharedMemory<T>();
  // Perform first level of reduction, reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int grid_size = block_size * gridDim.x;

  T my_sum = 0;

  // We reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger grid_size and therefore fewer elements per thread
  if (n_is_pow2) {
    unsigned int i = blockIdx.x * block_size * 2 + threadIdx.x;
    grid_size = grid_size << 1;

    while (i < n) {
      my_sum += g_idata[i];
      // ensure we don't read out of bounds -- this is optimized away for
      // powerOf2 sized arrays
      if ((i + block_size) < n) {
        my_sum += g_idata[i + block_size];
      }
      i += grid_size;
    }
  } else {
    unsigned int i = blockIdx.x * block_size + threadIdx.x;
    while (i < n) {
      my_sum += g_idata[i];
      i += grid_size;
    }
  }

  // each thread puts its local sum into shared memory
  sdata[tid] = my_sum;
  __syncthreads();

  // do reduction in shared mem
  if ((block_size >= 512) && (tid < 256)) {
    sdata[tid] = my_sum = my_sum + sdata[tid + 256];
    __syncthreads();
  }

  if ((block_size >= 256) && (tid < 128)) {
    sdata[tid] = my_sum = my_sum + sdata[tid + 128];
    __syncthreads();
  }

  if ((block_size >= 128) && (tid < 64)) {
    sdata[tid] = my_sum = my_sum + sdata[tid + 64];
  }

  __syncthreads();

  if (tid < 32) {
    // Fetch final intermediate sum from 2nd warp
    if (block_size >= 64) my_sum += sdata[tid + 32];
    // Reduce final warp using shuffle
    for (int offset = 1; offset < 32; offset <<= 1) {
        my_sum += __shfl_down_sync(0xffffffff, my_sum, offset);
    }
  }

  // write result for this block to global mem
  if (tid == 0) atomicAdd(&g_odata[0], my_sum);
}


void GPUReductionSum(float* h_in, float* h_out, int size)
{
    if (size < 1)
    {
        *h_out = 0;
        return;
    };
    float *d_in, *d_out;
    int block_size = 128;
    int num_blocks = (size+block_size-1) / block_size;
    unsigned int n_is_pow2 = !(size & (size-1));
    unsigned int smemsize = block_size * sizeof(float);
    hipMalloc(&d_in, size*sizeof(float));
    hipMalloc(&d_out, 1*sizeof(float));
    hipMemcpy(d_in, h_in, size*sizeof(float), hipMemcpyHostToDevice);
    if (n_is_pow2)
    {
        switch(block_size)
        {
            case 64:
                SumKernel<float,64, true><<< num_blocks, block_size, smemsize >>>(d_in, d_out, size); break;
            case 128:
                SumKernel<float,128, true><<< num_blocks, block_size, smemsize >>>(d_in, d_out, size); break;
            case 256:
                SumKernel<float,256, true><<< num_blocks, block_size, smemsize >>>(d_in, d_out, size); break;
            case 512:
                SumKernel<float,512, true><<< num_blocks, block_size, smemsize >>>(d_in, d_out, size); break;
        }
    }
    else
    {
        switch(block_size)
        {
            case 64:
                SumKernel<float,64, false><<< num_blocks, block_size, smemsize >>>(d_in, d_out, size); break;
            case 128:
                SumKernel<float,128, false><<< num_blocks, block_size, smemsize >>>(d_in, d_out, size); break;
            case 256:
                SumKernel<float,256, false><<< num_blocks, block_size, smemsize >>>(d_in, d_out, size); break;
            case 512:
                SumKernel<float,512, false><<< num_blocks, block_size, smemsize >>>(d_in, d_out, size); break;
        }
    };
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, 1*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
}

void MultiGPUReductionSum(float* h_in, float* h_out, int size)
{
    if (size < 1)
    {
        *h_out = 0;
        return;
    };
    omp_set_dynamic(0);
    int num_gpus;
    hipGetDeviceCount(&num_gpus);
    omp_set_num_threads(num_gpus);
    float* tmp_h_out = (float*)malloc(num_gpus * sizeof(float));
    #pragma omp parallel num_threads(num_gpus)
    {
        int thread_id = omp_get_thread_num();
        int partition_size = size / num_gpus;
        int index = thread_id*partition_size;
        if (thread_id == num_gpus-1)
        {
            partition_size = size - index;
        }
        hipSetDevice(thread_id);
        float *d_in, *d_out;
        int block_size = 128; //TODO: dynamically choose block_size instead of static
        int num_blocks = (partition_size+block_size-1) / block_size;
        unsigned int n_is_pow2 = !(partition_size & (partition_size-1));
        unsigned int smemsize = block_size * sizeof(float);
        hipError_t status_in = hipMalloc((void**)&d_in, partition_size*sizeof(float));
        if (status_in != hipSuccess)
        {
            printf("Error allocating pinned host memory -- in\n");
        }
        hipError_t status_out = hipMalloc((void**)&d_out, 1*sizeof(float));
        if (status_out != hipSuccess)
        {
            printf("Error allocating pinned host memory -- out\n");
        }
        hipMemset(d_out, 0, 1*sizeof(float));

        hipMemcpy(d_in, h_in+index, partition_size*sizeof(float), hipMemcpyHostToDevice);
        if (n_is_pow2)
        {
            switch(block_size)
            {
                case 64:
                    SumKernel<float, 64, true><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
                case 128:
                    SumKernel<float, 128, true><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
                case 256:
                    SumKernel<float, 256, true><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
                case 512:
                    SumKernel<float, 512, true><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
            }
        }
        else
        {
            switch(block_size)
            {
                case 64:
                    SumKernel<float, 64, false><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
                case 128:
                    SumKernel<float, 128, false><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
                case 256:
                    SumKernel<float, 256, false><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
                case 512:
                    SumKernel<float, 512, false><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
            }
        };
        hipDeviceSynchronize();//hipStreamSynchronize(); if multiple streams per device
        hipMemcpy(&tmp_h_out[thread_id], d_out, 1*sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_in);
        hipFree(d_out);
    }
    for (int i=1; i< num_gpus; i++)
    {
        tmp_h_out[0] += tmp_h_out[i];
    }
    *h_out = tmp_h_out[0];
    free(tmp_h_out);
}

// MultiGPUReductionSum2 relies on *loads supplied from LoadBalancer (HeteroNaiveSum.cpp and LoadBalancer.cpp)
void MultiGPUReductionSum2(float* h_in, float* h_out, int* loads, int size)
{
    if (size < 1)
    {
        *h_out = 0;
        return;
    };
    omp_set_dynamic(0);
    int num_gpus;
    hipGetDeviceCount(&num_gpus);
    omp_set_num_threads(num_gpus);
    float* tmp_h_out = (float*)calloc(num_gpus,sizeof(float));
    int partition_size;
    int index;
    #pragma omp parallel num_threads(num_gpus) private(partition_size, index)
    {
        int thread_id = omp_get_thread_num();
        partition_size = *(loads+thread_id);
        index = thread_id*partition_size;
        if(thread_id = num_gpus-1)
        {
            partition_size = size - index;
        }
        hipSetDevice(thread_id);
        float *d_in, *d_out;
        int block_size = 128; //TODO: dynamically choose block_size instead of static
        int num_blocks = (partition_size+block_size-1) / block_size;
        unsigned int n_is_pow2 = !(partition_size & (partition_size-1));
        unsigned int smemsize = block_size * sizeof(float);
        hipError_t status_in = hipMalloc((void**)&d_in, partition_size*sizeof(float));
        if (status_in != hipSuccess)
        {
            printf("Error allocating pinned host memory -- in\n");
        }
        hipError_t status_out = hipMalloc((void**)&d_out, 1*sizeof(float));
        if (status_out != hipSuccess)
        {
            printf("Error allocating pinned host memory -- out\n");
        }
        hipMemset(d_out, 0, 1*sizeof(float));

        hipMemcpy(d_in, h_in, partition_size*sizeof(float), hipMemcpyHostToDevice);
        if (n_is_pow2)
        {
            switch(block_size)
            {
                case 64:
                    SumKernel<float, 64, true><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
                case 128:
                    SumKernel<float, 128, true><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
                case 256:
                    SumKernel<float, 256, true><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
                case 512:
                    SumKernel<float, 512, true><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
            }
        }
        else
        {
            switch(block_size)
            {
                case 64:
                    SumKernel<float, 64, false><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
                case 128:
                    SumKernel<float, 128, false><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
                case 256:
                    SumKernel<float, 256, false><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
                case 512:
                    SumKernel<float, 512, false><<< num_blocks, block_size, smemsize >>>(d_in, d_out, partition_size); break;
            }
        };
        hipDeviceSynchronize();//hipStreamSynchronize(); if multiple streams per device
        hipMemcpy(&tmp_h_out[thread_id], d_out, 1*sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_in);
        hipFree(d_out);
    }
    for (int i=1; i< num_gpus; i++)
    {
        tmp_h_out[0] += tmp_h_out[i];
    }
    *h_out = tmp_h_out[0];
    free(tmp_h_out);
}

/*
    Pinned memory vs pageable memory overview:

    PROS: 1. Overlap copy operations and kernel launches
          2. Faster host2device and device2host transfers (limited by PCIe x(lanes) speed)
          3. Does not need to allocate on device, can directly read/write from/to pinned memory e.g. no need for
             hipMalloc(d_in...)
          4. Inter-GPU communication can benefit greatly as CPU is bypassed (direct memory access). Make sure inter-gpu
             communications are batched to saturation threshold if message size is small.

    CONS: 1. OS needs to allocate large contiguous memory block on RAM. Can easily run out of host RAM.
          2. Number 1 above is slower than allocating pageable memory.
          3. Unless pinned memory is reused multiple times, the performance gain of faster h2d/d2h transfers is dwarfed.

    OVERALL: For reduction sum: Because the data buffer is not reused multiple times and no inter-GPU communication is
          required, use of pinned memory results in major performance loss. Opt to use pageable memory for reduction
          sum.
*/
void GPUReductionSumPinned(float* h_in, float* h_out, int size)
{
    if (size < 1)
    {
        *h_out = 0;
        return;
    };
    float *dh_pinned_in, *dh_pinned_out;
    int block_size = 128;
    int num_blocks = (size+block_size-1) / block_size;
    unsigned int n_is_pow2 = !(size & (size-1));
    unsigned int smemsize = block_size * sizeof(float);
    hipError_t status_in = hipHostMalloc((void**)&dh_pinned_in, size*sizeof(float));
    if (status_in != hipSuccess)
    {
        printf("Error allocating pinned host memory -- in\n");
    }
    hipError_t status_out = hipHostMalloc((void**)&dh_pinned_out, 1*sizeof(float));
    if (status_out != hipSuccess)
    {
        printf("Error allocating pinned host memory -- out\n");
    }
    hipMemsetAsync(dh_pinned_out, 0, 1*sizeof(float));

    hipMemcpyAsync(dh_pinned_in, h_in, size*sizeof(float), hipMemcpyHostToDevice);
    if (n_is_pow2)
    {
        switch(block_size)
        {
            case 64:
                SumKernel<float, 64, true><<< num_blocks, block_size, smemsize >>>(dh_pinned_in, dh_pinned_out, size); break;
            case 128:
                SumKernel<float, 128, true><<< num_blocks, block_size, smemsize >>>(dh_pinned_in, dh_pinned_out, size); break;
            case 256:
                SumKernel<float, 256, true><<< num_blocks, block_size, smemsize >>>(dh_pinned_in, dh_pinned_out, size); break;
            case 512:
                SumKernel<float, 512, true><<< num_blocks, block_size, smemsize >>>(dh_pinned_in, dh_pinned_out, size); break;
        }
    }
    else
    {
        switch(block_size)
        {
            case 64:
                SumKernel<float, 64, false><<< num_blocks, block_size, smemsize >>>(dh_pinned_in, dh_pinned_out, size); break;
            case 128:
                SumKernel<float, 128, false><<< num_blocks, block_size, smemsize >>>(dh_pinned_in, dh_pinned_out, size); break;
            case 256:
                SumKernel<float, 256, false><<< num_blocks, block_size, smemsize >>>(dh_pinned_in, dh_pinned_out, size); break;
            case 512:
                SumKernel<float, 512, false><<< num_blocks, block_size, smemsize >>>(dh_pinned_in, dh_pinned_out, size); break;
        }
    };
    hipDeviceSynchronize();
    hipMemcpy(h_out, dh_pinned_out, 1*sizeof(float), hipMemcpyDeviceToHost);
    hipHostFree(dh_pinned_in);
    hipHostFree(dh_pinned_out);
}

void MultiGPUReductionSumPinned(float* h_in, float* h_out, int size)
{
    if (size < 1)
    {
        *h_out = 0;
        return;
    };
    omp_set_dynamic(0);
    int num_gpus;
    hipGetDeviceCount(&num_gpus);
    omp_set_num_threads(num_gpus);
    float *pinned_in, *pinned_out;

    // hipHostMallocPortable flag will make the memory portable across all contexts
    hipError_t status_in = hipHostAlloc((void**)&pinned_in, size*sizeof(float), hipHostMallocPortable);
    if (status_in != hipSuccess)
    {
        printf("Error allocating pinned host memory -- in\n");
    }
    hipError_t status_out = hipHostAlloc((void**)&pinned_out, num_gpus*sizeof(float), hipHostMallocPortable);
    if (status_out != hipSuccess)
    {
        printf("Error allocating pinned host memory -- out\n");
    }
    hipMemsetAsync(pinned_out, 0, 1*sizeof(float));

    hipMemcpy(pinned_in, h_in, size*sizeof(float), hipMemcpyHostToDevice);

    #pragma omp parallel num_threads(num_gpus)
    {
        int thread_id = omp_get_thread_num();
        int partition_size = size / num_gpus;
        int index = thread_id*partition_size;
        if (thread_id == num_gpus-1)
        {
            partition_size = size - index;
        }
        hipSetDevice(thread_id);
        int block_size = 128; //TODO: dynamically choose block_size instead of static
        int num_blocks = (partition_size+block_size-1) / block_size;
        unsigned int n_is_pow2 = !(partition_size & (partition_size-1));
        unsigned int smemsize = block_size * sizeof(float);
        if (n_is_pow2)
        {
            switch(block_size)
            {
                case 64:
                    SumKernel<float, 64, true><<< num_blocks, block_size, smemsize>>>
                    (pinned_in+index,pinned_out+thread_id, partition_size); break;
                case 128:
                    SumKernel<float, 128, true><<< num_blocks, block_size, smemsize >>>
                    (pinned_in+index, pinned_out+thread_id, partition_size); break;
                case 256:
                    SumKernel<float, 256, true><<< num_blocks, block_size, smemsize >>>
                    (pinned_in+index, pinned_out+thread_id, partition_size); break;
                case 512:
                    SumKernel<float, 512, true><<< num_blocks, block_size, smemsize >>>
                    (pinned_in+index, pinned_out+thread_id, partition_size); break;
            }
        }
        else
        {
            switch(block_size)
            {
                case 64:
                    SumKernel<float, 64, false><<< num_blocks, block_size, smemsize >>>
                    (pinned_in+index, pinned_out+thread_id, partition_size); break;
                case 128:
                    SumKernel<float, 128, false><<< num_blocks, block_size, smemsize >>>
                    (pinned_in+index, pinned_out+thread_id, partition_size); break;
                case 256:
                    SumKernel<float, 256, false><<< num_blocks, block_size, smemsize >>>
                    (pinned_in+index, pinned_out+thread_id, partition_size); break;
                case 512:
                    SumKernel<float, 512, false><<< num_blocks, block_size, smemsize >>>
                    (pinned_in+index, pinned_out+thread_id, partition_size); break;
            }
        };
        hipDeviceSynchronize();//hipStreamSynchronize(); if multiple streams per device
    }
    for (int i=1; i< num_gpus; i++)
    {
        pinned_out[0] += pinned_out[i];
    }
    *h_out = pinned_out[0];
    hipFree(pinned_in);
    hipFree(pinned_out);
}