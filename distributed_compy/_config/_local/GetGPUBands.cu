#include "GetGPUBands.h"
// #include <stdio.h>

int GetNumGPUs()
{
    int n_devices;
    hipGetDeviceCount(&n_devices);
    return n_devices;
}

float* GetGPUBands(int n_devices)
{
    float* gpu_bands = (float*)malloc(n_devices*sizeof(float));
    for (int i = 0; i < n_devices; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        int clock_rate = prop.memoryClockRate;
        int bus_width = prop.memoryBusWidth/8;
        float device_band = (2.0*clock_rate*(bus_width) / 1000000);
        gpu_bands[i] = (2.0*clock_rate*(bus_width) / 1000000);
    }
    return gpu_bands;
}